#include "hip/hip_runtime.h"
#include "KernelLauncher.cuh"

__device__
float gpu_smoothstep(float edge0, float edge1, float x) {
    float t = fmaxf(0.0f, fminf(1.0f, (x - edge0) / (edge1 - edge0)));
    return t * t * (3.0f - 2.0f * t);
}

__global__
void Kernel(uchar* data, double x, double y, int width, int height){
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    const int offset = r*width + c; 
    
    if ((c >= width) || (r >= height)) return ;
   
    const float distance = sqrt((c-x)*(c-x) + (r-y)*(r-y)) ;
    const float flashlightRadius = sqrt(x*x + y*y) ;
    //  smooth interpolation function (e.g., smoothstep) to calculate intensity
    float intensity = gpu_smoothstep(-1.0f, 1.0f, 1.0f - (distance / flashlightRadius));
    
    intensity = fminf(fmaxf(intensity, 0.0f), 1.0f);
    data[4*offset+1] = 255 ;
    data[4*offset+2] = 255;
    data[4*offset+3] = 255 ;
    data[4*offset+4] = 255*intensity ;
   

}



void KernelLauncher::Launcher(uchar *data_host, double &x, double& y){
    std::cout << "(x,y) :(" << x << "," << y <<")"<<std::endl;
    //sizeof(uchar) = 1
    //p_nbchannels*p_height*p_width*sizeof(uchar) = 4*w*h*1

    hipMalloc((void**)&data_device, p_nbchannels*p_height*p_width*sizeof(uchar));
    
    dim3 grid(p_width, p_height) ;
    Kernel<<<grid, 1>>>(data_device, x, y, p_width, p_height) ;
    hipMemcpy(data_host, data_device,p_nbchannels*p_height*p_width*sizeof(uchar), hipMemcpyDeviceToHost ) ;
}
 
KernelLauncher::KernelLauncher(const int& width, const int& height) 
: p_width(width), p_height(height) {
    p_data = new uchar[p_nbchannels*p_height*p_width*sizeof(uchar)] ;

}


KernelLauncher::~KernelLauncher(){

    delete [] p_data ;
    hipFree(data_device) ;

}

